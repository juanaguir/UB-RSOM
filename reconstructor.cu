#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <windows.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <time.h>

__global__ void pBP(float *recc, int *lineind, float *datacq, int * guid, int lz, int ly, int Sx,  int blockx, int blocky, int nblockx, int nblocky){
 

int jj = blockIdx.x;
int j = threadIdx.x;

		 jj=jj+blocky*nblocky;
		 j=j+blockx*nblockx;
		
			for(int i =0; i < (lz-1); i++ ) {
				for(int ii =  guid[i]-1 ; ii <  guid[i+1] ; ii++ ) {				
					recc[(j*(lz-1)*ly)+(jj*(lz-1))+i] = recc[(j*(lz-1)*ly)+(jj*(lz-1))+i] + datacq [int( lineind[ii] ) - 1 + j + jj * Sx ];
	               
																   }
				  		
			
		
}

 

#define N 512
void main( void ) {
	hipDeviceReset();  

	// managing and gathering host memory //
FILE *fpunt;	// pointer for write and read from harddrive
	 double *Nsx; // Number of elements from the acquired data
	 double *datacq; //sinogram
	 float *datacqf; //sinogram in float type precision (less memory better for GPU, althoung oduble is not that bad...). Could come like this from MATLAB. I KNOW.
	 double *lineind; // addresses of the voxels to be summed in a line of voxels following the trandsucer axis
	 int *lineindint; // addresses of the voxels to be summed in int type (less memory)
	 double *guid;    // guide to read the voxels addresses.
	 int *guidint;
	 
	 int nblockx; int nblocky; // number of blocks in x direction, y direction.  
	 int blockx; int blocky;  // block being reconstructed.
	 int lbx; int lby;  // Number of lines per blocks (number of parralell threads= lbx*lby)  
	float *recc; // reconstruction
	 double *lxyz; //vector containing the size of the array of addresses and the size of the acquired data
	 int Sx; int Sy; int Sz; // Size of the acquired data 
	 int lx; int ly; int lz; // number of veoxels in the reconstuction grid
	
	 int lindex; // lenght of the array containing the memory addresses of the voxels to be sum,
	 int *A; // variable to guide the parallel for in the cuda function
	 Nsx = (double *)malloc(sizeof(double) * 3);
	lxyz= (double *)malloc(sizeof(double) * 4);
	int zlim; // limit for each iteration of the reconstructor 
	int dum; // "helper" variable
	double dumm; // "helper" variable
	dum=3;
    
	 printf("Loading data and copying it to the GPU memory\n");

	//               Gathering the size of the acquired data    //
	fpunt = fopen("path", "rb");
    fread(Nsx, sizeof(double),dum, fpunt); 
    fclose(fpunt);
	

     Sx=int(Nsx[0]); Sy=int(Nsx[1]); Sz=int(Nsx[2]);
	 

	datacq = (double *)calloc(sizeof(double),Sx*Sy*Sz);
	 fpunt = fopen("path", "rb");
    fread(datacq, sizeof(double),Sx*Sy*Sz, fpunt); 
    fclose(fpunt);
	 datacqf = (float *)calloc(sizeof(float),Sx*Sy*Sz);
	 for(int i=0; i<Sx*Sy*Sz; i++){
	 datacqf[i]=float(datacq[i]);
	 }
	  free(datacq);
	 //Gathering the size of the vector containing the memory addresses and acquired data without zeros 
	
	dum=4;
	fpunt = fopen("path", "rb");
    fread(lxyz, sizeof(double),dum, fpunt); 
    fclose(fpunt);
	 
    ly=int(lxyz[0]); lx=int(lxyz[1]); lz=int(lxyz[2]); lindex=int(lxyz[3]);
   	 
	//Gathering the vector of addresses

	lineind = (double *)malloc(sizeof(double)*lindex); //acquired data
	fpunt = fopen("path", "rb");
    fread(lineind, sizeof(double),lindex, fpunt); 
    fclose(fpunt);
	 lineindint = (int *)calloc(sizeof(int),lindex);
	 for(int i=0; i<lindex; i++){
	 lineindint[i]=int(lineind[i]);
	 }
	 free(lineind);

    //Gathering the guide of the vector of addresses
    guid = (double*)malloc(sizeof(double)*lz); //acquired data
	fpunt = fopen("path", "rb");
     fread(guid, sizeof(double),lz, fpunt); 
    fclose(fpunt);
	 guidint = (int*)malloc(sizeof(int)*lz); //acquired data
	for(int i=0;i<lz;i++){
		guidint[i]=int(guid[i]);
	}
	 free(guid);
     // allocating memory for the reconstruction
	recc = ( float* ) calloc( sizeof( float), ( lz-1 ) * lx * ly );

	
// changing mclasses to svae memmoery 
	

// managing and gathering device memory//
int *devA; 
float *devrecc; 
int *devlineind; 
float *devdatacq; 
int *devguid;  

hipMalloc( (void**)&devrecc, sizeof(float)* ( lz-1 ) * lx * ly);
hipMemcpy( devrecc, recc,sizeof(float)* ( lz-1 ) * lx * ly, hipMemcpyHostToDevice);
hipMalloc( (void**)&devlineind, sizeof(int)*lindex );
hipMemcpy( devlineind, lineindint,sizeof(int)*lindex, hipMemcpyHostToDevice);
hipMalloc( (void**)&devdatacq, sizeof(float)*Sx*Sy*Sz);
hipMemcpy( devdatacq, datacqf,sizeof(float)*Sx*Sy*Sz, hipMemcpyHostToDevice);
hipMalloc( (void**)&devguid, sizeof(int)*lz);
hipMemcpy( devguid, guidint,sizeof(int)*lz, hipMemcpyHostToDevice);


//************************ setting the number of lines to be reconstructed in parallel (lbx*lby) *************************//
lbx=50;  
lby=50;  
//***********************************************************************************************************************//

nblockx= int(floor(double(lx)/double(lbx))); // number of blocks in x direction
nblocky= int(floor(double(ly)/double(lby))); // number of blocks in y direction


dim3 blocks(lbx, 1); // conditioning cuda memory blocks
dim3 grids(lby, 1);  // 

//dim3 blocks(lx, 1); // conditioning cuda meory blocks
//dim3 grids(ly, 1);

int dumblock=0; // dummy variable to count the numer of blocks already reconstructed
printf("Reconstructing\n");
//for(blockx=0; blockx<nblockx; blockx++) {
blockx=0;
blocky=0;
clock_t tic = clock();
for(blockx=0; blockx<nblockx; blockx++) {
	for(blocky=0;blocky<nblocky; blocky++) {
	pBP<<< grids,blocks >>>(devrecc, devlineind, devdatacq, devguid, lz,ly, Sx, blockx,blocky,lbx,lby);
    hipDeviceSynchronize();
    dumblock+=1;
    printf("block %d out of %d reconstructed \n", dumblock,nblockx*nblocky);
	}
}
clock_t toc = clock();
 printf("Elapsed: %f seconds\n", (double)(toc - tic) / CLOCKS_PER_SEC);

 printf("reconstruction finished, saving data\n");
hipMemcpy( recc, devrecc, sizeof(float)* ( lz-1 ) * lx * ly, hipMemcpyDeviceToHost);
hipFree( devrecc);
hipFree( devlineind);
hipFree( devdatacq);
hipFree( devguid);


double* reccd = ( double* ) calloc( sizeof( double ), ( lz-1 ) * lx * ly );
for(int i=0; i<( lz-1 ) * lx * ly ;i++){
reccd[i]=double(recc[i]);
}

fpunt = fopen("path","w b");

if (fpunt == NULL)
{
printf("The file did not open");
}

fwrite (reccd, sizeof(double),lx*ly*(lz-1),fpunt);
fclose(fpunt);
 printf("Fisnished! press any key");
 getchar();

}